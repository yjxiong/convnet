#include "hip/hip_runtime.h"
/*
Copyright (c) 2009,2010, Volodymyr Mnih
All rights reserved.

Redistribution and use in source and binary forms, with or without modification, are permitted provided that 
the following conditions are met:

    * Redistributions of source code must retain the above copyright notice, this list of conditions and the 
following disclaimer.
    * Redistributions in binary form must reproduce the above copyright notice, this list of conditions and 
the following disclaimer in the documentation and/or other materials provided with the distribution.
    * Neither the name of the <ORGANIZATION> nor the names of its contributors may be used to endorse or 
promote products derived from this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED 
WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A 
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY 
DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, 
PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR 
OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH 
DAMAGE.
*/
#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>
#include <math.h>
#include "rnd_multipliers_32bit.h"
#include "cudamat_kernels.cuh"

#include <iostream>
using namespace std;

extern "C" {

#include "cudamat.cuh"

/* ------------------------------ CUBLAS init/shutdown ------------------------------ */

inline bool check_cublas_error() {
    hipblasStatus_t status = cublasGetError();
    return status != HIPBLAS_STATUS_SUCCESS;
}

inline bool checkCUDAError() {
    hipError_t err = hipGetLastError();

    if (hipSuccess != err)
        printf("%s\n", hipGetErrorString( err));
    return hipSuccess != err;
}

const char* get_last_cuda_error() {
    hipError_t err = hipGetLastError();

    return hipGetErrorString( err);
}

int cublas_init() {
    cublasInit();
    if (check_cublas_error())
        return CUBLAS_ERROR;
    else
        return 0;
}

int cublas_shutdown() {
    cublasShutdown();
    hipDeviceReset();
    return 0;
}

int cuda_record_event(hipEvent_t* t) {
  hipError_t err = hipEventRecord(*t, 0);
  if (hipSuccess != err) {
    printf("%s\n", hipGetErrorString( err));
  }
  return hipSuccess != err;
}

int cuda_synchronize_event(hipEvent_t* t) {
  hipError_t err = hipEventSynchronize(*t);
  //hipError_t err = hipStreamWaitEvent(NULL, *t, 0);
  if (hipSuccess != err) {
    printf("%s\n", hipGetErrorString( err));
  }
  return hipSuccess != err;
}

int cuda_create_event(hipEvent_t* t) {
  hipError_t err = hipEventCreateWithFlags(t, hipEventBlockingSync);
  //hipError_t err = hipEventCreate(t);
  if (hipSuccess != err) {
    printf("%s\n", hipGetErrorString( err));
  }
  return hipSuccess != err;
}

int cuda_set_device(int deviceId) {
    hipSetDevice(deviceId);
    
    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

bool cuda_is_fermi(int deviceId) {
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, deviceId);
  return prop.major >= 2;
}

int cuda_set_P2P(int gpu1, int gpu2) {
  bool is_fermi = cuda_is_fermi(gpu1) && cuda_is_fermi(gpu2);
  
  int access2from1, access1from2;

  hipDeviceCanAccessPeer(&access2from1, gpu1, gpu2);
  hipDeviceCanAccessPeer(&access1from2, gpu2, gpu1);

  //printf("%d can access %d : %d\n ", gpu1, gpu2, access2from1);
  //printf("%d can access %d : %d\n ", gpu2, gpu1, access1from2);

  bool same_complex = false;
  if(access2from1==1 && access1from2==1) same_complex = true;

  if(is_fermi && same_complex) {
    hipSetDevice(gpu1);
    hipDeviceEnablePeerAccess(gpu2, 0); //second argument is flags
    hipSetDevice(gpu2);
    hipDeviceEnablePeerAccess(gpu1, 0); //second argument is flags
    return 0;
  } else {
    return CUDA_ERROR;
  }
}

int init_random(rnd_struct* rnd_state, int seed, const char* cudamatpath) {
    unsigned int * host_mults;
    host_mults = (unsigned int*)malloc(NUM_RND_STREAMS * sizeof(unsigned int));

    for (int i = 0; i < NUM_RND_STREAMS; i++) {
      host_mults[i] = _rand_words[i];
    }

    cublasAlloc(NUM_RND_STREAMS, sizeof(unsigned int), (void**)&rnd_state->dev_mults);
    cublasAlloc(NUM_RND_STREAMS, sizeof(unsigned long long), (void**)&rnd_state->dev_words);
    hipblasSetVector(NUM_RND_STREAMS, sizeof(unsigned int), host_mults, 1, rnd_state->dev_mults, 1);
    free(host_mults);
    //hipMalloc((void **)&rnd_state->dev_mults, NUM_RND_STREAMS * sizeof(unsigned int));
    //hipMalloc((void **)&rnd_state->dev_words, NUM_RND_STREAMS * sizeof(unsigned long long));
    //hipMemcpy(rnd_state->dev_mults, host_mults, NUM_RND_STREAMS * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    kSeedRandom<<<NUM_RND_BLOCKS, NUM_RND_THREADS_PER_BLOCK>>>(rnd_state->dev_mults, rnd_state->dev_words, seed);

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

// Allocates and gives up ownership of pointer. Caller must free.
int get_rnd_state(rnd_struct* rnd_state, unsigned long long* host_words_out, int *size_out) {
  *size_out = NUM_RND_STREAMS;
  host_words_out = (unsigned long long*)malloc(NUM_RND_STREAMS * sizeof(unsigned int));
  if (host_words_out == NULL) {
    return ERROR_GENERIC;  // Out of memory.
  }
  hipblasGetVector(NUM_RND_STREAMS, sizeof(unsigned long long), rnd_state->dev_words, 1, host_words_out, 1);
  if (check_cublas_error())
    return CUBLAS_ERROR;
  else
     return 0;
}

int init_random_from_state(rnd_struct* rnd_state, unsigned long long* host_words, char* cudamatpath) {
    unsigned int * host_mults;
    host_mults = (unsigned int*)malloc(NUM_RND_STREAMS * sizeof(unsigned int));
    FILE * pFile;

    if (cudamatpath == NULL) {
      pFile = fopen ("/u/nitish/deepnet/cudamat/rnd_multipliers_32bit.txt","r");
    } else {
      pFile = fopen (cudamatpath,"r");
    }
    if (pFile == NULL) {
      printf("Error: Missing rnd_multipliers_32bit.txt file\n");
      return 1;
    }

    for (int i = 0; i < NUM_RND_STREAMS; i++) {
      int r = fscanf (pFile, "%u", &host_mults[i]);
      if (r != 1) return ERROR_GENERIC;
    }
    fclose (pFile);

    cublasAlloc(NUM_RND_STREAMS, sizeof(unsigned int), (void**)&rnd_state->dev_mults);
    cublasAlloc(NUM_RND_STREAMS, sizeof(unsigned long long), (void**)&rnd_state->dev_words);
    hipblasSetVector(NUM_RND_STREAMS, sizeof(unsigned int), host_mults, 1, rnd_state->dev_mults, 1);
    hipblasSetVector(NUM_RND_STREAMS, sizeof(unsigned int), host_words, 1, rnd_state->dev_words, 1);
    free(host_mults);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

/* ------------------------------ Utility routines ------------------------------ */

int get_leading_dimension(cudamat* mat) {
    return mat->is_trans ? mat->size[1] : mat->size[0];
}

int get_nonleading_dimension(cudamat* mat) {
    return mat->is_trans ? mat->size[0] : mat->size[1];
}

void set_transpose(cudamat* mat, int is_trans) {
    mat->is_trans = is_trans;
}

inline char get_transpose_char(cudamat* mat) {
    return mat->is_trans ? 't' : 'n';
}

void cuda_sync_threads() {
    hipDeviceSynchronize();
}

/* ------------------------------ Allocating/moving data ------------------------------ */

int allocate_device_memory(cudamat* mat) {
    int len = mat->size[0]*mat->size[1];

    hipblasStatus_t stat;

    stat = cublasAlloc(len, sizeof(mat->data_device[0]), (void**)&mat->data_device);

    if (stat != HIPBLAS_STATUS_SUCCESS || check_cublas_error()) {
        checkCUDAError();
        return CUBLAS_ERROR;
    }

    mat->on_device = 1;
    return 0;
}

int allocate_device_memory_bbox(cudamat_bbox* mat) {
    int size = mat->size;
    int numboxes = mat->numboxes;

    hipblasStatus_t stat;

    stat = cublasAlloc(size, sizeof(int), (void**)&mat->data_device.seg);
    if (stat != HIPBLAS_STATUS_SUCCESS || check_cublas_error()) {
        checkCUDAError();
        return CUBLAS_ERROR;
    }
    stat = cublasAlloc(numboxes, sizeof(int), (void**)&mat->data_device.labels);
    if (stat != HIPBLAS_STATUS_SUCCESS || check_cublas_error()) {
        checkCUDAError();
        return CUBLAS_ERROR;
    }
    stat = cublasAlloc(4 * numboxes, sizeof(int), (void**)&mat->data_device.boxes);
    if (stat != HIPBLAS_STATUS_SUCCESS || check_cublas_error()) {
        checkCUDAError();
        return CUBLAS_ERROR;
    }

    mat->on_device = 1;
    return 0;
}

int allocate_device_memory_sparse(cudamat_sparse* mat) {
    int nnz = mat->nnz, rows = mat->size[0];

    hipblasStatus_t stat;

    stat = cublasAlloc(nnz, sizeof(mat->data_device.data[0]), (void**)&mat->data_device.data);
    if (stat != HIPBLAS_STATUS_SUCCESS || check_cublas_error()) {
        checkCUDAError();
        return CUBLAS_ERROR;
    }

    stat = cublasAlloc(nnz, sizeof(mat->data_device.indices[0]), (void**)&mat->data_device.indices);
    if (stat != HIPBLAS_STATUS_SUCCESS || check_cublas_error()) {
        checkCUDAError();
        return CUBLAS_ERROR;
    }

    stat = cublasAlloc(rows + 1, sizeof(mat->data_device.indptr[0]), (void**)&mat->data_device.indptr);
    if (stat != HIPBLAS_STATUS_SUCCESS || check_cublas_error()) {
        checkCUDAError();
        return CUBLAS_ERROR;
    }

    mat->on_device = 1;
    return 0;
}

int copy_to_host_slice(cudamat* mat, int start, int end) {
    if (start >= end || end > mat->size[1])
      return ERROR_GENERIC;

    int len = mat->size[0] * (end - start);
    int offset = mat->size[0] * start;

    if (mat->on_device) {
        hipblasGetVector(len, sizeof(mat->data_host[0]), mat->data_device + offset, 1, mat->data_host + offset, 1);

        if (check_cublas_error())
            return CUBLAS_ERROR;
    } else
       return ERROR_NOT_ON_DEVICE;
 
    return 0;
}

int copy_to_host(cudamat* mat) {
    int len = mat->size[0]*mat->size[1];

    if (mat->on_device) {
        hipblasGetVector(len, sizeof(mat->data_host[0]), mat->data_device, 1, mat->data_host, 1);

        if (check_cublas_error())
            return CUBLAS_ERROR;
    } else
       return ERROR_NOT_ON_DEVICE;
 
    return 0;
}

int copy_bbox_to_host(cudamat_bbox* mat) {
  if (mat->on_device) {
    hipblasGetVector(mat->size, sizeof(int), mat->data_device.seg, 1, mat->data_host.seg, 1);
    hipblasGetVector(mat->numboxes, sizeof(int), mat->data_device.labels, 1, mat->data_host.labels, 1);
    hipblasGetVector(4 * mat->numboxes, sizeof(int), mat->data_device.boxes, 1, mat->data_host.boxes, 1);
    if (check_cublas_error()) return CUBLAS_ERROR;
  } else {
    return ERROR_NOT_ON_DEVICE;
  }
  return 0;
}
int copy_to_device_slice(cudamat* mat, const int start, const int end) {
    if (end <= start || end > mat->size[1])
      return ERROR_GENERIC;

    int len = mat->size[0] * (end - start);
    int err_code = 0;
    int offset = mat->size[0] * start;
    //if (!mat->owns_data)
    //    return VIEW_ERROR;

    if (!mat->on_device) {
        err_code = allocate_device_memory(mat);
        if (err_code)
            return err_code;
    }
    hipblasSetVector(len, sizeof(mat->data_host[0]), mat->data_host + offset, 1, mat->data_device + offset, 1);

    if (check_cublas_error())
        return CUBLAS_ERROR;

    return 0;
}



int copy_to_device(cudamat* mat) {
    int len = mat->size[0]*mat->size[1];
    int err_code = 0;

    //if (!mat->owns_data)
    //    return VIEW_ERROR;

    if (!mat->on_device) {
        err_code = allocate_device_memory(mat);
        if (err_code)
            return err_code;
    }

    hipblasSetVector(len, sizeof(mat->data_host[0]), mat->data_host, 1, mat->data_device, 1);
    
    if (check_cublas_error())
        return CUBLAS_ERROR;

    return 0;
}

int copy_bbox_to_device(cudamat_bbox* mat) {
    int size = mat->size;
    int numboxes = mat->numboxes;
    int err_code = 0;

    //if (!mat->owns_data)
    //    return VIEW_ERROR;

    if (!mat->on_device) {
        err_code = allocate_device_memory_bbox(mat);
        if (err_code)
            return err_code;
    }

    hipblasSetVector(size, sizeof(int), mat->data_host.seg, 1, mat->data_device.seg, 1);
    if (check_cublas_error())
        return CUBLAS_ERROR;
    hipblasSetVector(numboxes, sizeof(int), mat->data_host.labels, 1, mat->data_device.labels, 1);
    if (check_cublas_error())
        return CUBLAS_ERROR;
    hipblasSetVector(4 * numboxes, sizeof(int), mat->data_host.boxes, 1, mat->data_device.boxes, 1);
    if (check_cublas_error())
        return CUBLAS_ERROR;

    return 0;
}

int copy_sparse_to_device(cudamat_sparse* mat) {
    int len = mat->nnz, rows = mat->size[0];
    int err_code = 0;

    //if (!mat->owns_data)
    //    return VIEW_ERROR;

    if (!mat->on_device) {
        err_code = allocate_device_memory_sparse(mat);
        if (err_code)
            return err_code;
    }

    hipblasSetVector(len, sizeof(mat->data_host.data[0]), mat->data_host.data, 1, mat->data_device.data, 1);
    if (check_cublas_error())
        return CUBLAS_ERROR;

    hipblasSetVector(len, sizeof(mat->data_host.indices[0]), mat->data_host.indices, 1, mat->data_device.indices, 1);
    if (check_cublas_error())
        return CUBLAS_ERROR;

    hipblasSetVector(rows + 1, sizeof(mat->data_host.indptr[0]), mat->data_host.indptr, 1, mat->data_device.indptr, 1);
    if (check_cublas_error())
        return CUBLAS_ERROR;

    return 0;
}

// mat 1 : source
// mat 2 : dest
int copy_on_device(cudamat* mat1, cudamat* mat2) {
    int len = mat1->size[0]*mat1->size[1];

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    hipMemcpy(mat2->data_device, mat1->data_device, len * sizeof(float), hipMemcpyDefault);
    //hipblasScopy(len, mat1->data_device, 1, mat2->data_device, 1);

    if (check_cublas_error())
        return CUBLAS_ERROR;
    else
        return 0;
}
int copy_on_device_p2p_async(cudamat* src, cudamat* dst, int src_dev, int dst_dev) {
    int len = src->size[0]*src->size[1];

    if (src->size[0] != dst->size[0] || src->size[1] != dst->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    hipMemcpyPeerAsync(dst->data_device, dst_dev, src->data_device, src_dev, len * sizeof(float));

    if (check_cublas_error())
        return CUBLAS_ERROR;
    else
        return 0;
}


int get_row_slice(cudamat* source, cudamat* target, unsigned int start, unsigned int end) {
    int height = source->size[0];
    int width = source->size[1];

    if ((end - start) != target->size[0] || source->size[1] != target->size[1] || start >= end || end > height)
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    dim3 kernelBlockGrid((int)ceil((end - start)/32.), (int)ceil(width/32.), 1);
    dim3 kernelBlockDim(32, 1, 1);

    kGetRowSlice<<<kernelBlockGrid,kernelBlockDim>>>(source->data_device, target->data_device, start, end, width, height);

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

int set_row_slice(cudamat* source, cudamat* target, unsigned int start, unsigned int end) {
    int height = target->size[0];
    int width = target->size[1];

    if ((end - start) != source->size[0] || source->size[1] != target->size[1] || start >= end || end > height)
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    dim3 kernelBlockGrid((int)ceil((end - start)/32.), (int)ceil(width/32.), 1);
    dim3 kernelBlockDim(32, 1, 1);

    kSetRowSlice<<<kernelBlockGrid,kernelBlockDim>>>(source->data_device, target->data_device, start, end, width, height);

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

int copy_transpose_big_matrix(cudamat* source, cudamat* target) {
    unsigned int height = source->size[0];
    unsigned int width = source->size[1];

    if (source->size[0] != target->size[1] || source->size[1] != target->size[0])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kTransposeBig<<< NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK >>>(target->data_device, source->data_device, height, width);

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}


int copy_transpose(cudamat* source, cudamat* target) {
    unsigned int height = source->size[0];
    unsigned int width = source->size[1];

    if (source->size[0] != target->size[1] || source->size[1] != target->size[0])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    // setup execution parameters
    unsigned int grid_x = height / COPY_BLOCK_SIZE;
    if (height % COPY_BLOCK_SIZE)
        grid_x++;

    unsigned int grid_y = width / COPY_BLOCK_SIZE;
    if (width % COPY_BLOCK_SIZE)
        grid_y++;

    dim3 grid(grid_x, grid_y, 1);
    dim3 threads(COPY_BLOCK_SIZE, COPY_BLOCK_SIZE, 1);

    kTranspose<<< grid, threads >>>(target->data_device, source->data_device, height, width);

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

int free_device_memory(cudamat* mat) {
    if (mat->owns_data && mat->on_device) {
        hipblasStatus_t stat;

        stat = cublasFree(mat->data_device);
        mat->on_device = 0;

        if (stat != HIPBLAS_STATUS_SUCCESS || check_cublas_error())
            return CUBLAS_ERROR;
    }

    return 0;
}

int free_device_memory_bbox(cudamat_bbox* mat) {
    if (mat->on_device) {
        hipblasStatus_t stat;

        stat = cublasFree(mat->data_device.seg);
        stat = cublasFree(mat->data_device.labels);
        stat = cublasFree(mat->data_device.boxes);
        mat->on_device = 0;

        if (stat != HIPBLAS_STATUS_SUCCESS || check_cublas_error())
            return CUBLAS_ERROR;
    }

    return 0;
}


int set_shape(cudamat* mat, unsigned int m, unsigned int n) {

    mat->size[0] = m;
    mat->size[1] = n;

    return 0;
}


int reshape(cudamat* mat, int m, int n) {
    if (m < 0 && n < 0)
        return ERROR_GENERIC;
    if (m < 0)
        m = (mat->size[0] * mat->size[1]) / n;
    if (n < 0)
        n = (mat->size[0] * mat->size[1]) / m;

    if (mat->size[0] * mat->size[1] != m * n)
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    mat->size[0] = m;
    mat->size[1] = n;

    return 0;
}

int get_slice(cudamat* source, cudamat* target, unsigned int first_col, unsigned int last_col) {
    if (source->is_trans)
        return ERROR_TRANSPOSED;

    if (!source->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (last_col > source->size[1] || (first_col >= last_col))
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    int num_rows = source->size[0];

    target->data_host = source->data_host + first_col * num_rows;
    target->data_device = source->data_device + first_col * num_rows;
    target->on_device = 1;
    target->on_host = 0;
    target->size[0] = source->size[0];
    target->size[1] = last_col - first_col;
    target->is_trans = 0;
    target->owns_data = 0;

    return 0;
}

int get_vector_slice(cudamat* source, cudamat* target, unsigned int first_ind, unsigned int last_ind) {
    // source must be a vector.
    if (source->size[0] > 1 && source->size[1] > 1)
        return ERROR_GENERIC;

    if (source->is_trans)
        return ERROR_TRANSPOSED;

    if (!source->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (first_ind >= last_ind)
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    int num_rows = source->size[0];

    target->data_host = 0;
    target->data_device = source->data_device + first_ind * num_rows;
    target->on_device = 1;
    target->on_host = 0;
    target->is_trans = 0;
    target->owns_data = 0;

    if (source->size[0] > 1) {
        if (last_ind > source->size[0])
            return ERROR_INCOMPATIBLE_DIMENSIONS;

        target->size[0] = last_ind - first_ind;
        target->size[1] = 1;
    } else {
        if (last_ind > source->size[1])
            return ERROR_INCOMPATIBLE_DIMENSIONS;

        target->size[0] = 1;
        target->size[1] = last_ind - first_ind;
    }

    return 0;
}

/* ------------------------------ Initialization routines ------------------------------ */

void init_from_array(cudamat* mat, float* data, int m, int n) {
    mat->data_host = data;
    mat->size[0] = m;
    mat->size[1] = n;
    mat->on_device = 0;
    mat->on_host = 1;
    mat->is_trans = 0;
    mat->owns_data = 1;
}

void init_from_sparse_array(cudamat_sparse* mat, float* data, int* indices, int* indptr, int m, int n, int nnz) {
    mat->data_host.data = data;
    mat->data_host.indices = indices;
    mat->data_host.indptr = indptr;
    mat->size[0] = m;
    mat->size[1] = n;
    mat->on_device = 0;
    mat->on_host = 1;
    mat->is_trans = 0;
    mat->owns_data = 1;
    mat->nnz = nnz;
}


void set_on_device(cudamat* mat) {
  mat->on_device = 1;
}

int init_empty(cudamat* mat, int m, int n) {
    mat->size[0] = m;
    mat->size[1] = n;
    mat->on_device = 0;
    mat->on_host = 0;
    mat->is_trans = 0;
    mat->owns_data = 1;

    return allocate_device_memory(mat);
}

/* ------------------------------ Random number generation ------------------------------ */
int fill_with_rand(rnd_struct* rnd_state, cudamat* mat) {
    int len = mat->size[0] * mat->size[1];

    if (!mat->on_device)
        return ERROR_NOT_ON_DEVICE;

    kRandomUniform<<<NUM_RND_BLOCKS,NUM_RND_THREADS_PER_BLOCK>>>(rnd_state->dev_mults, rnd_state->dev_words, mat->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

int fill_with_randn(rnd_struct* rnd_state, cudamat* mat) {
    int len = mat->size[0] * mat->size[1];

    if (!mat->on_device)
        return ERROR_NOT_ON_DEVICE;

    kRandomGaussian<<<NUM_RND_BLOCKS,NUM_RND_THREADS_PER_BLOCK>>>(rnd_state->dev_mults, rnd_state->dev_words, mat->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

int sample_bernoulli(rnd_struct* rnd_state, cudamat* mat, cudamat* target) {
    int len = mat->size[0] * mat->size[1];
    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    if (!mat->on_device)
        return ERROR_NOT_ON_DEVICE;

    kSampleBernoulli<<<NUM_RND_BLOCKS,NUM_RND_THREADS_PER_BLOCK>>>(rnd_state->dev_mults, rnd_state->dev_words, mat->data_device, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}
int sample_bernoulli_tanh(rnd_struct* rnd_state, cudamat* mat, cudamat* target) {
    int len = mat->size[0] * mat->size[1];
    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    if (!mat->on_device)
        return ERROR_NOT_ON_DEVICE;

    kSampleBernoulliTanh<<<NUM_RND_BLOCKS,NUM_RND_THREADS_PER_BLOCK>>>(rnd_state->dev_mults, rnd_state->dev_words, mat->data_device, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}
int sample_poisson(rnd_struct* rnd_state, cudamat* mat, cudamat* target) {
    int len = mat->size[0] * mat->size[1];
    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    if (!mat->on_device)
        return ERROR_NOT_ON_DEVICE;

    kSamplePoisson<<<NUM_RND_BLOCKS,NUM_RND_THREADS_PER_BLOCK>>>(rnd_state->dev_mults, rnd_state->dev_words, mat->data_device, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}
int sample_gaussian(rnd_struct* rnd_state, cudamat* mat, cudamat* target, float mult) {
    int len = mat->size[0] * mat->size[1];
    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    if (!mat->on_device)
        return ERROR_NOT_ON_DEVICE;

    kSampleGaussian<<<NUM_RND_BLOCKS,NUM_RND_THREADS_PER_BLOCK>>>(rnd_state->dev_mults, rnd_state->dev_words, mat->data_device, target->data_device, len, mult);

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

int perturb_energy(rnd_struct* rnd_state, cudamat* mat, cudamat* target) {
    int len = mat->size[0] * mat->size[1];
    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    if (!mat->on_device)
        return ERROR_NOT_ON_DEVICE;

    kPerturbEnergy<<<NUM_RND_BLOCKS,NUM_RND_THREADS_PER_BLOCK>>>(rnd_state->dev_mults, rnd_state->dev_words, mat->data_device, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

int perturb_prob(rnd_struct* rnd_state, cudamat* mat, cudamat* target) {
    int len = mat->size[0] * mat->size[1];
    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    if (!mat->on_device)
        return ERROR_NOT_ON_DEVICE;

    kPerturbProb<<<NUM_RND_BLOCKS,NUM_RND_THREADS_PER_BLOCK>>>(rnd_state->dev_mults, rnd_state->dev_words, mat->data_device, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

int dropout(rnd_struct* rnd_state, cudamat* mat, float dropprob, float val, float scale) {
    int len = mat->size[0] * mat->size[1];

    if (!mat->on_device)
        return ERROR_NOT_ON_DEVICE;

    kRandomDropout<<<NUM_RND_BLOCKS,NUM_RND_THREADS_PER_BLOCK>>>(rnd_state->dev_mults, rnd_state->dev_words, mat->data_device, len, dropprob, val, scale);

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

int gaussian_dropout(rnd_struct* rnd_state, cudamat* mat, float scale) {
    int len = mat->size[0] * mat->size[1];

    if (!mat->on_device)
        return ERROR_NOT_ON_DEVICE;

    kRandomGaussianDropout<<<NUM_RND_BLOCKS,NUM_RND_THREADS_PER_BLOCK>>>(rnd_state->dev_mults, rnd_state->dev_words, mat->data_device, len, scale);

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}


/* ------------------------------ Algebraic operations ------------------------------ */

int add_col_vec(cudamat* mat, cudamat* vec, cudamat* target) {
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !vec->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (mat->size[0] != vec->size[0] || vec->size[1] != 1 ||
        mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    unsigned int num_blocks = DIVUP((w * h), (NUM_VECTOR_OP_LOOPS_PER_THREAD * NUM_VECTOR_OP_THREADS_PER_BLOCK));
    num_blocks = MIN(NUM_VECTOR_OP_BLOCKS, num_blocks);
    kAddColVector<<<num_blocks,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, vec->data_device, target->data_device, w, h);

    if (checkCUDAError()) {
        return CUDA_ERROR;
    }

    return 0;
}

int add_col_mult(cudamat* mat, cudamat* vec, cudamat* target, float mult) {
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !vec->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (mat->size[0] != vec->size[0] || vec->size[1] != 1 ||
        mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    unsigned int num_blocks = DIVUP((w * h), (NUM_VECTOR_OP_LOOPS_PER_THREAD * NUM_VECTOR_OP_THREADS_PER_BLOCK));
    num_blocks = MIN(NUM_VECTOR_OP_BLOCKS, num_blocks);
    kAddColMult<<<num_blocks,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, vec->data_device, target->data_device, mult, w, h);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int add_to_each_pixel(cudamat* mat1, cudamat* mat2, cudamat* target, float mult) {
    unsigned int h = mat1->size[0],
                 w = mat1->size[1],
                 num_colors = mat2->size[1];

    if (!mat1->on_device || !mat2->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans || mat2->is_trans)
        return ERROR_TRANSPOSED;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] % mat2->size[1] != 0 ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kAddToEachPixel<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, mult, w, h, w / num_colors);

    if (checkCUDAError()) {
        return CUDA_ERROR;
    }

    return 0;
}


int mult_diagonal_scalar(cudamat* mat, float val, cudamat* target) {
    unsigned int w = mat->size[1];

    if (!mat->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kMultDiagonalScalar<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, val, target->data_device, w);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}


int add_diagonal_scalar(cudamat* mat, float val, cudamat* target) {
    unsigned int w = mat->size[1];

    if (!mat->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kAddDiagonalScalar<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, val, target->data_device, w);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}


int mult_diagonal(cudamat* mat, cudamat* vec, cudamat* target) {
    unsigned int w = mat->size[1];

    if (!mat->on_device || !vec->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (mat->size[0] != vec->size[1] * vec->size[0] ||
        mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kMultDiagonal<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, vec->data_device, target->data_device, w);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}


int add_diagonal(cudamat* mat, cudamat* vec, cudamat* target) {
    unsigned int w = mat->size[1];

    if (!mat->on_device || !vec->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (mat->size[0] != vec->size[1] * vec->size[0] ||
        mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kAddDiagonal<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, vec->data_device, target->data_device, w);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}


int add_row_mult(cudamat* mat, cudamat* vec, cudamat* target, float mult) {
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !vec->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (mat->size[1] != vec->size[1] || vec->size[0] != 1 ||
        mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    unsigned int num_blocks = DIVUP((w * h), (NUM_VECTOR_OP_LOOPS_PER_THREAD * NUM_VECTOR_OP_THREADS_PER_BLOCK));
    num_blocks = MIN(NUM_VECTOR_OP_BLOCKS, num_blocks);
    kAddRowMult<<<num_blocks,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, vec->data_device, target->data_device, mult, w, h);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int add_row_vec(cudamat* mat, cudamat* vec, cudamat* target) {
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !vec->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (mat->size[1] != vec->size[1] || vec->size[0] != 1 ||
        mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    unsigned int num_blocks = DIVUP((w * h), (NUM_VECTOR_OP_LOOPS_PER_THREAD * NUM_VECTOR_OP_THREADS_PER_BLOCK));
    num_blocks = MIN(NUM_VECTOR_OP_BLOCKS, num_blocks);
    kAddRowVector<<<num_blocks,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, vec->data_device, target->data_device, w, h);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int mult_by_col_vec(cudamat* mat, cudamat* vec, cudamat* target) {
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !vec->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (mat->size[0] != vec->size[0] || vec->size[1] != 1 ||
        mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kMultByColVector<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, vec->data_device, target->data_device, w, h);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int mult_by_row_vec(cudamat* mat, cudamat* vec, cudamat* target) {
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !vec->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (mat->size[1] != vec->size[1] || vec->size[0] != 1 ||
        mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kMultByRowVector<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, vec->data_device, target->data_device, w, h);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int div_by_col_vec(cudamat* mat, cudamat* vec, cudamat* target) {
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !vec->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (mat->size[0] != vec->size[0] || vec->size[1] != 1 ||
        mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kDivByColVector<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, vec->data_device, target->data_device, w, h);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int div_by_row_vec(cudamat* mat, cudamat* vec, cudamat* target) {
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !vec->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (mat->size[1] != vec->size[1] || vec->size[0] != 1 ||
        mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kDivByRowVector<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, vec->data_device, target->data_device, w, h);


    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}
int less_than_eq(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kLessThanEq<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int less_than(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kLessThan<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}
int less_than_eq_scalar(cudamat* mat, float val, cudamat* target) {
    int len = mat->size[0]*mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans != target->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kLessThanEqScalar<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, val, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}


int less_than_scalar(cudamat* mat, float val, cudamat* target) {
    int len = mat->size[0]*mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans != target->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kLessThanScalar<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, val, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int greater_than_eq(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kGreaterThanEq<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int greater_than(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kGreaterThan<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int upper_bound(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kUpperBound<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}


int lower_bound(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kLowerBound<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}
int greater_than_eq_scalar(cudamat* mat, float val, cudamat* target) {
    int len = mat->size[0]*mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans != target->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kGreaterThanEqScalar<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, val, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int greater_than_scalar(cudamat* mat, float val, cudamat* target) {
    int len = mat->size[0]*mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans != target->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kGreaterThanScalar<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, val, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}
int upper_bound_scalar(cudamat* mat, float val, cudamat* target) {
    int len = mat->size[0]*mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans != target->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kUpperBoundScalar<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, val, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int upper_bound_mod_scalar(cudamat* mat, float val, cudamat* target) {
    int len = mat->size[0]*mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans != target->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kUpperBoundModScalar<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, val, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}


int lower_bound_scalar(cudamat* mat, float val, cudamat* target) {
    int len = mat->size[0]*mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans != target->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kLowerBoundScalar<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, val, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int max_by_axis(cudamat* mat, cudamat* target, int axis) {
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (axis == 0) {
        if (target->size[0] != 1 || target->size[1] != mat->size[1])
            return ERROR_INCOMPATIBLE_DIMENSIONS;

        int shared_mem_size = 32 * sizeof(float) ;
        int w1 = floor(sqrt(w));
        int w2 = w / w1 + (w % w1 == 0 ? 0 : 1);
        dim3 gridDim(w1, w2, 1);
        kMaxColumnwise<<<gridDim, 32, shared_mem_size>>>(mat->data_device, target->data_device, w, h);

    } else
        return ERROR_UNSUPPORTED;

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int choose_max_and_accumulate(cudamat* mat, cudamat* acc) {
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !acc->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (acc->size[0] != mat->size[0] || acc->size[1] != mat->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

        int w1 = floor(sqrt(w));
        int w2 = w / w1 + (w % w1 == 0 ? 0 : 1);
        dim3 gridDim(w1, w2, 1);
    kChooseMaxAndAccumulate<<<gridDim,32>>>(mat->data_device, acc->data_device, w, h);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}
int choose_max_by_axis(cudamat* mat, cudamat* target, int axis) {
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (axis == 0) {
        if (target->size[0] != mat->size[0] || target->size[1] != mat->size[1])
            return ERROR_INCOMPATIBLE_DIMENSIONS;

        int shared_mem_size = 32 * sizeof(float) ;
        int w1 = floor(sqrt(w));
        int w2 = w / w1 + (w % w1 == 0 ? 0 : 1);
        dim3 gridDim(w1, w2, 1);
        kChooseMaxColumnwise<<<gridDim, 32, shared_mem_size>>>(mat->data_device, target->data_device, w, h);

    } else
        return ERROR_UNSUPPORTED;

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}
int argmax_by_axis(cudamat* mat, cudamat* target, int axis) {
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (axis == 0) {
        if (target->size[0] != 1 || target->size[1] != mat->size[1])
            return ERROR_INCOMPATIBLE_DIMENSIONS;

        int w1 = floor(sqrt(w));
        int w2 = w / w1 + (w % w1 == 0 ? 0 : 1);
        dim3 gridDim(w1, w2, 1);
        kArgMaxColumnwise<<<gridDim,32>>>(mat->data_device, target->data_device, w, h);

    } else
        return ERROR_UNSUPPORTED;

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}
int sqsum_by_axis(cudamat* mat, cudamat* target, int axis, float mult, float p) {
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (axis == 0) {
        if (target->size[0] != 1 || target->size[1] != mat->size[1])
            return ERROR_INCOMPATIBLE_DIMENSIONS;

        int shared_mem_size = 32 * sizeof(float) ;
        int w1 = floor(sqrt(w));
        int w2 = (w + w1 - 1) / w1;
        dim3 gridDim(w1, w2, 1);
        kSqSumColumnwise<<<gridDim, 32, shared_mem_size>>>(mat->data_device, target->data_device, w, h, mult, p);
    } else if (axis == 1) {
        if (target->size[1] != 1 || target->size[0] != mat->size[0])
            return ERROR_INCOMPATIBLE_DIMENSIONS;

        int shared_mem_size = 32 * sizeof(float) ;
        int h1 = floor(sqrt(h));
        int h2 = (h + h1 - 1) / h1;
        dim3 gridDim(h1, h2, 1);
        kSqSumRowwise<<<gridDim, 32, shared_mem_size>>>(mat->data_device, target->data_device, w, h, mult, p);
    } else
        return ERROR_UNSUPPORTED;

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int normlimit_by_axis(cudamat* mat, cudamat* target, int axis,
                                   float norm, int constraint) {
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (target->size[0] != mat->size[0] || target->size[1] != mat->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    int shared_mem_size = 32 * sizeof(float) ;
    if (axis == 0) {
        int w1 = floor(sqrt(w));
        int w2 = DIVUP(w, w1);
        dim3 gridDim(w1, w2, 1);
        kNormLimitColumnwise<<<gridDim,32, shared_mem_size>>>(mat->data_device, target->data_device, norm, w, h, constraint);
    } else {
        int h1 = floor(sqrt(h));
        int h2 = DIVUP(h, h1);
        dim3 gridDim(h1, h2, 1);
        kNormLimitRowwise<<<gridDim,32, shared_mem_size>>>(mat->data_device, target->data_device, norm, w, h, constraint);
    }
    if (checkCUDAError())
        return CUDA_ERROR;
    return 0;
}


int sign(cudamat* mat, cudamat* target) {
    int len = mat->size[0]*mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans != target->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kSign<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}
int apply_cos(cudamat* mat, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kApplyCos<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}
int apply_sin(cudamat* mat, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kApplySin<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int apply_sigmoid(cudamat* mat, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kApplySigmoid<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int apply_tanh(cudamat* mat, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kApplyTanh<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int apply_abs(cudamat* mat, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kApplyAbs<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int apply_log_1_plus_exp(cudamat* mat, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kApplyLog1PlusExp<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

// target = 2 / (1 + exp(-mat * lambda)) - 1
int apply_relu_squash(cudamat* mat, cudamat* target, float lambda) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kSquashRelu<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len, lambda);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int apply_log(cudamat* mat, cudamat* target, float tiny) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kLog<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len, tiny);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int apply_exp(cudamat* mat, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kExp<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}
int apply_ceil(cudamat* mat, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kCeil<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}
int apply_floor(cudamat* mat, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kFloor<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}



int apply_sqrt(cudamat* mat, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kSqrt<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int apply_pow(cudamat* mat, float pow, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kPow<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, pow, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int apply_pow_matrix(cudamat* mat, cudamat* pow, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    if (mat->size[0] != pow->size[0] || mat->size[1] != pow->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kPowMatrix<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, pow->data_device, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int compute_cross_entropy(cudamat* dist1, cudamat* dist2, cudamat* target, float tiny) {
    unsigned int len = dist1->size[0] * dist1->size[1];

    if (!dist1->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (dist1->size[0] != target->size[0] || dist1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    if (dist1->size[0] != dist2->size[0] || dist1->size[1] != dist2->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kCrossEntropy<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(dist1->data_device, dist2->data_device, target->data_device, len, tiny);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}
int compute_cross_entropy_bernoulli(cudamat* mat, cudamat* pow, cudamat* target, float tiny) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    if (mat->size[0] != pow->size[0] || mat->size[1] != pow->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kCrossEntropyBernoulli<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, pow->data_device, target->data_device, len, tiny);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}
int correct_preds(cudamat* mat, cudamat* pow, cudamat* target, float cutoff) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    if (mat->size[0] != pow->size[0] || mat->size[1] != pow->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kCorrectPreds<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, pow->data_device, target->data_device, len, cutoff);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int reciprocal(cudamat* mat, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kReciprocal<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

// target = beta * target + alpha * mat * mat2
int dot(cudamat* mat1, cudamat* mat2, cudamat* target, float beta, float alpha) {
    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (get_leading_dimension(mat1) != get_leading_dimension(target) ||
        get_nonleading_dimension(mat2) != get_nonleading_dimension(target) ||
        get_nonleading_dimension(mat1) != get_leading_dimension(mat2)) {
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    }
    int m = get_leading_dimension(mat1),
        k = get_leading_dimension(mat2),
        n = get_nonleading_dimension(mat2);

    hipblasSgemm(get_transpose_char(mat1), get_transpose_char(mat2), 
                m, n, k,
                alpha, mat1->data_device, mat1->size[0],
                mat2->data_device, mat2->size[0],
                beta, target->data_device, target->size[0]);

    if (check_cublas_error())
        return CUBLAS_ERROR;

    return 0;
}

int sparse_dot(cudamat_sparse* mat1, cudamat* mat2, cudamat* target, float beta, float alpha) {
    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;
    int m = mat1->size[0],
        k = mat1->size[1],
        k2 = mat2->size[0],
        n = mat2->size[1];

    if (k != k2) {
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    }
    unsigned int grid_x = m / COPY_BLOCK_SIZE;
    if (m % COPY_BLOCK_SIZE)
        grid_x++;

    unsigned int grid_y = n / COPY_BLOCK_SIZE;
    if (n % COPY_BLOCK_SIZE)
        grid_y++;

    dim3 grid(grid_x, grid_y, 1);
    dim3 threads(COPY_BLOCK_SIZE, COPY_BLOCK_SIZE, 1);

    kSparseDot<<<grid, threads>>>(m, n, k, mat1->data_device.data,
        mat1->data_device.indptr,
        mat1->data_device.indices,
        mat2->data_device, target->data_device, beta, alpha);
    if (check_cublas_error())
        return CUBLAS_ERROR;

    return 0;
}


float vdot(cudamat* mat1, cudamat* mat2, int* err_code) {
    int len = mat1->size[0]*mat1->size[1];
    float res;

    if (!mat1->on_device || !mat2->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans) {
        *err_code = ERROR_TRANSPOSEDNESS;
        return 0;
    }

    if (mat2->size[0] * mat2->size[1] != len) {
        *err_code = ERROR_INCOMPATIBLE_DIMENSIONS;
        return 0;
    }

    res = hipblasSdot(len, mat1->data_device, 1, mat2->data_device, 1);

    if (check_cublas_error()) {
        *err_code = CUBLAS_ERROR;
        return -1.;
    } else {
        *err_code = 0;
        return res;
    }
}

/* Perform the operation mat1 = mat1 + alpha * mat2. mat1 and mat2 must
   have the same transposedness. */
int add_mult(cudamat* mat1, cudamat* mat2, float alpha) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    hipblasSaxpy(len, alpha, mat2->data_device, 1, mat1->data_device, 1);

    if (check_cublas_error())
        return CUBLAS_ERROR;

    return 0;
}
int add_mult_sign(cudamat* mat1, cudamat* mat2, float mult) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kAddMultSign<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, len, mult);

    if (check_cublas_error())
        return CUBLAS_ERROR;

    return 0;
}


int add_elementwise(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kAdd<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int subtract_elementwise(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kSubtract<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int divide_elementwise(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kDivide<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

/* Elementwise multiplication of 2 matrices */
int mult_elementwise(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kMult<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int apply_sin_deriv(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kSinDeriv<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}
int apply_cos_deriv(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kCosDeriv<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}
int apply_logistic_deriv(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kLogisticDeriv<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

// mat1 - output of network
// mat2 - target
// out_grad - output gradient
int apply_logistic_grad(cudamat* mat1, cudamat* mat2, cudamat* out_grad) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device || !out_grad->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != out_grad->size[0] || mat1->size[1] != out_grad->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kLogisticGrad<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, out_grad->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

// mat1 - output of network
// mat2 - target
// out - .
int get_logistic_correct_normalized(cudamat* mat1, cudamat* mat2, cudamat* out) {

    if (!mat1->on_device || !mat2->on_device || !out->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != out->size[0] || 1 != out->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    int num_blocks = DIVUP(mat1->size[0], NUM_VECTOR_OP_THREADS_PER_BLOCK);
    kLogisticCorrectNormalized<<<num_blocks, NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, out->data_device, mat1->size[0], mat1->size[1]);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int apply_tanh_deriv(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kTanhDeriv<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int apply_rectified_linear_deriv(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kRectifiedLinearDeriv<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int apply_rectified_linear_smooth_deriv(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kRectifiedLinearSmoothDeriv<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}
int assign_scalar(cudamat* mat, float alpha) {
    int len = mat->size[0]*mat->size[1];

    if (!mat->on_device)
        return ERROR_NOT_ON_DEVICE;

    kAssignScalar<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, alpha, len);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int mult_by_scalar(cudamat* mat, float alpha, cudamat* target) {
    int len = mat->size[0]*mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kMultScalar<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, alpha, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int divide_by_scalar(cudamat* mat, float alpha, cudamat* target) {
    int len = mat->size[0]*mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kDivideScalar<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, alpha, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int add_scalar(cudamat* mat, float alpha, cudamat* target) {
    int len = mat->size[0]*mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kAddScalar<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, alpha, target->data_device, len);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

float euclid_norm(cudamat* mat, int* err_code) {
    int len = mat->size[0]*mat->size[1];

    float res =  hipblasSnrm2(len, mat->data_device, 1);

    if (!mat->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (check_cublas_error()) {
        *err_code = CUBLAS_ERROR;
        return -1.;
    } else {
        *err_code = 0;
        return res;
    }
}
int selectRows(cudamat* source, cudamat* target, cudamat* indices){
    const int nRetRows = indices->size[1];

    if (nRetRows==0) return 0;

    dim3 gridDim((nRetRows+31)/32);
    dim3 blockDim(32);

    kSelectRows<<<gridDim, blockDim>>>(source->data_device, target->data_device, indices->data_device, nRetRows, source->size[0], source->size[1]);

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}


int swapColumns(cudamat* source, cudamat* target, cudamat* indices1, cudamat* indices2){
    const int cols = indices1->size[1]*indices1->size[0],
                 h = source->size[0],
                 w = source->size[1];

    kSwapColumns<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(source->data_device, target->data_device, indices1->data_device, indices2->data_device, cols, w, h);

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

int shuffleColumns(cudamat* source, cudamat* rand_perm_indices) {
    const int h = source->size[0],
              w = source->size[1];
    if (rand_perm_indices->size[0] != 1 || rand_perm_indices->size[1] != w) {
      return ERROR_INCOMPATIBLE_DIMENSIONS;
    }

    kShuffleColumns<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(source->data_device, source->data_device, rand_perm_indices->data_device, w, h);

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

int setSelectedRows(cudamat* target, cudamat* source, cudamat* indices){
    const int nSetRows = indices->size[1];

    if (nSetRows==0)
        return 0;

    dim3 gridDim((nSetRows+31)/32);
    dim3 blockDim(32);

    kSetSelectedRows<<<gridDim, blockDim>>>(target->data_device, source->data_device, indices->data_device, nSetRows, target->size[0], target->size[1]);

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

int generate_translations_big_var_off(cudamat* source, cudamat* target, cudamat* off_x, cudamat* off_y, int source_w, int target_w, int num_channels) {
    dim3 kernelBlockGrid(source->size[1], 1, 1);
    dim3 kernelBlockDim(512, 1, 1);

    kGenerateTranslationsBigVarOff<<<kernelBlockGrid, kernelBlockDim>>>(source->data_device, target->data_device, off_x->data_device, off_y->data_device, source_w, target_w, num_channels);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int extract_patches(cudamat* images, cudamat* patches, cudamat* width_offset, cudamat* height_offset, cudamat* flip, int img_width, int img_height, int patch_width, int patch_height) {
  int num_images = images->size[1];
  int num_colors = images->size[0] / (img_width * img_height);

  if (patches->size[1]  != num_colors * patch_width * patch_height || patches->size[0] != num_images)
    return ERROR_INCOMPATIBLE_DIMENSIONS;

  if (width_offset->size[0] * width_offset->size[1] != num_images)
    return ERROR_INCOMPATIBLE_DIMENSIONS;

  if (height_offset->size[0] * height_offset->size[1] != num_images)
    return ERROR_INCOMPATIBLE_DIMENSIONS;

  if (flip->size[0] * flip->size[1] != num_images)
    return ERROR_INCOMPATIBLE_DIMENSIONS;

    unsigned int grid_x = patch_height / COPY_BLOCK_SIZE;
    if (patch_height % COPY_BLOCK_SIZE)
        grid_x++;

    unsigned int grid_y = patch_width / COPY_BLOCK_SIZE;
    if (patch_width % COPY_BLOCK_SIZE)
        grid_y++;

    dim3 grid(grid_x, grid_y, num_images);
    dim3 threads(COPY_BLOCK_SIZE, COPY_BLOCK_SIZE, num_colors);


  kExtractPatches2<<<grid, threads>>>(
      images->data_device, patches->data_device, width_offset->data_device,
      height_offset->data_device, flip->data_device, num_images, img_width, img_height,
      patch_width, patch_height, num_colors);
  //*/
  /*
  kExtractPatches<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(
      images->data_device, patches->data_device, indices->data_device, width_offset->data_device,
      height_offset->data_device, num_images, img_width, img_height,
      patch_width, patch_height, num_colors);
   */

  if (checkCUDAError())
    return CUDA_ERROR;
  return 0;
}

int blockify(cudamat* source, cudamat* target, int blocksize) {
    dim3 kernelBlockGrid(source->size[1], 1, 1);
    dim3 kernelBlockDim(512, 1, 1);
    kBlockify<<<kernelBlockGrid, kernelBlockDim>>>(source->data_device, target->data_device, source->size[0], blocksize);
    if (checkCUDAError())
        return CUDA_ERROR;
    return 0;
}


int softmax(cudamat* mat, cudamat* target) { 
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (target->size[0] != h || target->size[1] != w)
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    
    int shared_mem_size = 32 * sizeof(float) ;

    int w1 = floor(sqrt(w));
    int w2 = w / w1 + (w % w1 == 0 ? 0 : 1);
    dim3 gridDim(w1, w2, 1);
    kSoftMax<<<gridDim, 32, shared_mem_size>>>(mat->data_device, target->data_device, w, h);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int softmax_overwrite(cudamat* mat) { 
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    int shared_mem_size = 32 * sizeof(float) ; 
    int w1 = floor(sqrt(w));
    int w2 = w / w1 + (w % w1 == 0 ? 0 : 1);
    dim3 gridDim(w1, w2, 1);
    kSoftMaxOverwrite<<<gridDim, 32, shared_mem_size>>>(mat->data_device, w, h);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int softmax_row_major(cudamat* mat) {
    return softmax_row_major_multi(mat, mat->size[1]);
}

int softmax_row_major_multi(cudamat* mat, int numslices) {
    unsigned int len = mat->size[0] * mat->size[1];
    unsigned int h = len / numslices;

    if (len % numslices != 0)
      return ERROR_INCOMPATIBLE_DIMENSIONS;

    if (!mat->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    int shared_mem_size = 32 * sizeof(float) ; 
    int h1 = floor(sqrt(h));
    int h2 = h / h1 + (h % h1 == 0 ? 0 : 1);
    dim3 gridDim(h1, h2, 1);
    kSoftMaxRowMajor<<<gridDim, 32, shared_mem_size>>>(mat->data_device, numslices, h);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int apply_softmax_grad(cudamat* mat, cudamat* labels, cudamat* target) { 
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (target->size[0] != h || target->size[1] != w)
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    if (labels->size[0] != 1 || labels->size[1] != w)
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    
    kSoftMaxGrad<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, labels->data_device, target->data_device, w, h);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int apply_softmax_grad_CLS(cudamat* mat, cudamat_bbox* labels, cudamat* indices, cudamat* target) { 
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (target->size[0] != h || target->size[1] != w)
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    
    kSoftMaxGradCLS<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(
        mat->data_device, labels->data_device.labels, indices->data_device, target->data_device, w, h);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;

}
int apply_softmax_grad_row_major(cudamat* mat, cudamat* labels, cudamat* target) { 
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (target->size[0] != h || target->size[1] != w)
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    if (labels->size[0] * labels->size[1] != h)
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    
    kSoftMaxGradRowMajor<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, labels->data_device, target->data_device, w, h);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int apply_grad_bbox(
    cudamat* mat, cudamat_bbox* bbox, cudamat* indices, cudamat* width_offset,
    cudamat* height_offset, cudamat* target, int width, int height, int depth,
    float scale_width, float scale_height, int loss_function) {

    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !target->on_device || !bbox->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (target->size[0] != h || target->size[1] != w)
        return ERROR_INCOMPATIBLE_DIMENSIONS;
   
    if (loss_function == 0) {
      //int grid_y = DIVUP(height, COPY_BLOCK_SIZE);
      //int grid_x = DIVUP(width, COPY_BLOCK_SIZE) * h;
      dim3 grid(width, height, depth);
      dim3 threads(h, 1, 1);
      kBoundingBoxLogisticGrad<<<grid, threads>>>(
          mat->data_device, bbox->data_device.boxes, bbox->data_device.labels,
          bbox->data_device.seg, indices->data_device, width_offset->data_device,
          height_offset->data_device, h, width, height, depth, scale_width,
          scale_height, target->data_device);
   
    } else {
      kBoundingBoxSoftMaxGrad<<<NUM_VECTOR_OP_BLOCKS, NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(
          mat->data_device, bbox->data_device.boxes, bbox->data_device.labels,
          bbox->data_device.seg, indices->data_device, width_offset->data_device,
          height_offset->data_device, h, width, height, depth, scale_width,
          scale_height, target->data_device);
    }
    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}



int get_softmax_correct(cudamat* mat, cudamat* labels, cudamat* target) { 
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (target->size[0] != 1 || target->size[1] != w)
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    if (labels->size[0] != 1 || labels->size[1] != w)
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    
    int w1 = floor(sqrt(w));
    int w2 = w / w1 + (w % w1 == 0 ? 0 : 1);
    dim3 gridDim(w1, w2, 1);
    kSoftMaxCorrect<<<gridDim, 32>>>(mat->data_device, labels->data_device, target->data_device, w, h);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int get_softmax_correct_row_major(cudamat* mat, cudamat* labels, cudamat* target) { 
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (target->size[0] != h || target->size[1] != 1)
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    if (labels->size[0] * labels->size[1] != h)
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    
    int h1 = floor(sqrt(h));
    int h2 = h / h1 + (h % h1 == 0 ? 0 : 1);
    dim3 gridDim(h1, h2, 1);
    kSoftMaxCorrectRowMajor<<<gridDim, 32>>>(mat->data_device, labels->data_device, target->data_device, w, h);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int get_softmax_correct_CLS(cudamat* mat, cudamat_bbox* labels, cudamat* indices, cudamat* target) { 
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !target->on_device || !indices->on_device || !labels->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (target->size[0] * target->size[1] != h)
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    if (indices->size[0]  * indices->size[1] != h)
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    
    int h1 = floor(sqrt(h));
    int h2 = h / h1 + (h % h1 == 0 ? 0 : 1);
    dim3 gridDim(h1, h2, 1);
    kSoftMaxCorrectCLS<<<gridDim, 32>>>(mat->data_device, labels->data_device.labels, indices->data_device, target->data_device, w, h);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}



int get_softmax_correct_row_major_bbox(cudamat* mat, cudamat_bbox* bbox, cudamat* indices, cudamat* width_offset, cudamat* height_offset, cudamat* target, int width, int height, int depth, float scale_width, float scale_height) { 
    unsigned int h = mat->size[0] * width * height;

    if (!mat->on_device || !target->on_device || !bbox->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (target->size[0] * target->size[1] != h) {
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    }
    
    int h1 = floor(sqrt(h));
    int h2 = h / h1 + (h % h1 == 0 ? 0 : 1);
    dim3 gridDim(h1, h2, 1);

    kSoftMaxCorrectBoundingBox<<<gridDim, 32>>>(
        mat->data_device, bbox->data_device.boxes, bbox->data_device.labels,
        bbox->data_device.seg, indices->data_device, width_offset->data_device,
        height_offset->data_device, mat->size[0], width,
        height, depth, scale_width, scale_height, target->data_device);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int get_logistic_correct_row_major_bbox(cudamat* mat, cudamat_bbox* bbox, cudamat* indices, cudamat* width_offset, cudamat* height_offset, cudamat* target, int width, int height, int depth, float scale_width, float scale_height, float cutoff) { 
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !target->on_device || !bbox->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (target->size[0] != h || target->size[1] != w)
        return ERROR_INCOMPATIBLE_DIMENSIONS;
   
    int grid_y = DIVUP(height, COPY_BLOCK_SIZE);
    int grid_x = DIVUP(width, COPY_BLOCK_SIZE) * h;
    dim3 grid(grid_x, grid_y, depth);
    dim3 threads(COPY_BLOCK_SIZE, COPY_BLOCK_SIZE, 1);
    kLogisticCorrectBoundingBox<<<grid, threads>>>(
        mat->data_device, bbox->data_device.boxes, bbox->data_device.labels,
        bbox->data_device.seg, indices->data_device, width_offset->data_device,
        height_offset->data_device, h, width, height, depth, scale_width,
        scale_height, target->data_device, cutoff);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}



int accumulate_columns(cudamat* mat, cudamat* indices, cudamat* target, float mult, int avg) { 
    unsigned int h = mat->size[0],
                 w = mat->size[1],
                 w2 = target->size[1];

    if (!mat->on_device || !indices->on_device|| !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (target->size[0] != h)
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    if (indices->size[0] != 1 || indices->size[1] != w)
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    if (NUM_VECTOR_OP_THREADS_PER_BLOCK < w2)
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    
    kAccumulateColumns<<<h, NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, indices->data_device, target->data_device, w, w2, h, mult, avg);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int get_softmax_cross_entropy(cudamat* mat, cudamat* labels, cudamat* target, float tiny) { 
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (target->size[0] != 1 || target->size[1] != w)
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    if (labels->size[0] != 1 || labels->size[1] != w)
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    
    kSoftMaxCrossEntropy<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, labels->data_device, target->data_device, w, h, tiny);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

int get_softmax_cross_entropy_row_major(cudamat* mat, cudamat* labels, cudamat* target, float tiny) { 
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (target->size[0] != h || target->size[1] != 1)
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    if (labels->size[0] != h || labels->size[1] != 1)
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    
    kSoftMaxCrossEntropyRowMajor<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, labels->data_device, target->data_device, w, h, tiny);

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}


int expand(cudamat* source, cudamat* indices, cudamat* target){
    unsigned int h = source->size[0],
                 w = source->size[1],
                 w2 = target->size[1];

    if (!source->on_device || !indices->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (source->is_trans)
        return ERROR_TRANSPOSED;

    if (target->size[0] != h)
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    if (indices->size[0] != 1 || indices->size[1] != w2)
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kExpand<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(source->data_device, indices->data_device, target->data_device, h, w, w2);
    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}


int expand_and_add(cudamat* source, cudamat* mat, cudamat* indices, cudamat* target, float mult){
    unsigned int h = source->size[0],
                 w = source->size[1],
                 w2 = mat->size[1];

    if (!source->on_device || !mat->on_device || !indices->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (target->size[0] != h || target->size[1] != w)
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    if (indices->size[0] != 1 || indices->size[1] != w)
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    if (mat->size[0] != h)
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kExpandAndAdd<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(source->data_device, mat->data_device, indices->data_device, target->data_device, w, h, mult, w2);

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

int adagrad(cudamat* w, cudamat* grad, cudamat* sum_grad_sq, float decay, float epsilon) {
    int len = w->size[0] * w->size[1];
    int trans = w->is_trans;

    if (!w->on_device || !grad->on_device || !sum_grad_sq->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (trans != grad->is_trans || trans != sum_grad_sq->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (len != grad->size[0] * grad->size[1] || len != sum_grad_sq->size[0] * sum_grad_sq->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    kAdagrad<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(w->data_device, grad->data_device, sum_grad_sq->data_device, len, decay, epsilon);

    if (check_cublas_error())
        return CUBLAS_ERROR;

    return 0;
}
}
